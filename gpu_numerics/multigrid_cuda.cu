#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "multigrid_cuda.cuh"


//////////////////
// GPU KERNELS //
////////////////

/* Given in problem sets */
__device__
static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

/* Adapted from problem sets to find the maximum error using (simple)
   reduction technique. */
__global__
void computeErrorKernel( const float *u, const float *v, const int n, float *error) {

  extern __shared__ float emax[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  // reset error
  if (i == 0) error[0] = 0;

  // reset shared memory
  emax[tid] = 0;

  // Load to shared memory
  while (i < n+1) {
    float val = fabs(u[i] - v[i]);
    emax[tid] = fmax(val, emax[tid]);
    i += gridDim.x * blockDim.x;
  }
  __syncthreads();

  // Find max in block
  /* NOTE: this could be optimized further by unrolling the loop */
  for (unsigned int s = blockDim.x/2; s > 0; s = s / 2) {
    if (tid < s) {
      emax[tid] = fmax(emax[tid], emax[tid+s]);
    }
    __syncthreads();
  }

  // Find max across blocks
  if (tid == 0) atomicMax(error, emax[0]);

}

/* Jacobi solver is the basis of this whole project. Note that several
   different approaches were tested. This approach was taken in all
   subsequent kernels as well, where appropriate, but only the best
   was used. Here, all three were left, if interested. */

__global__
void jacobiCudaKernel( float *v, const float *vprev, const float *f, const int n) {

  extern __shared__ float vprev_sm[];
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  float y;
  float h2 = 1./n/n; // h * h


  // TAKE 1: serial version. Incredibly slow for obvious reasons, but
  // this approach was the first attempt in all my kernels to ensure
  // proper structure of my code.
  /*
  if (i == 0) {
    v[0] = 0;
    for (int j = 1; j<n; j++) {
      y = (vprev[j-1] + vprev[j+1] - f[j]*h2) / 2;
      v[j] = y * (1-WEIGHT) + WEIGHT * vprev[j];
    }
    v[n] = 0;
  }
  */


  // TAKE 2: shared memory. My motivation for this approach was that
  // several indices are needed to compute the desired value. It
  // seemed like a great idea (and was used on Lab 6, problem 1), but
  // the if statements do causes instances of warp
  // divergence. Additionally, reading the endpoints from global
  // memory are inefficient due to memory coalescing.
  /*
  while (i < n+1) {

    // load current data to shared memory
    unsigned int i_sm = threadIdx.x + 1;
    vprev_sm[i_sm] = vprev[i];
    if (threadIdx.x == 0) {
      if (i == 0) {
	vprev_sm[0] = 0;
      } else {
	vprev_sm[0] = vprev[i-1];
      }
    }
    if (threadIdx.x == blockDim.x-1) {
      if (i == n-1) {
	vprev_sm[i_sm+1] = 0;
      } else {
	vprev_sm[i_sm+1] = vprev[i+1];
      }
    }
    __syncthreads();
    
    // Solver
    y = (vprev_sm[i_sm-1] + vprev_sm[i_sm+1] - f[i]*h2) / 2;
    y *= (1-WEIGHT);
    y += WEIGHT * vprev_sm[i_sm];
    v[i] = y;

    // Increment index
    i += blockDim.x * gridDim.x;
  }
  */


  // TAKE 3: Read and write to global memory. Despite global memory
  // coalescing problems, this slightly beat my shared memory version.
  while (i < n) {

    if (i == 0) {
      v[0] = 0;
      v[n] = 0;
    } else {
      y = (vprev[i-1] + vprev[i+1] - f[i]*h2) / 2;
      y *= (1-WEIGHT);
      y += WEIGHT * vprev[i];
      v[i] = y;
      
    }

    // Increment index
    i += blockDim.x * gridDim.x;
  }
  
}

// Computes residual, r = Av - f.
__global__
void residualCudaKernel(float *r, const float *v, const float *f, const int n) {

  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  float n2 = (float) n*n;

  // Load global memory to hide latency. It did not make a big impact,
  // but was worth testing.
  while (i < n) {
    if (i == 0) {
      float v0 = v[0];
      float vn = v[n];
      float f0 = f[0];
      float fn = f[n];
      r[0] = v0 - f0;
      r[n] = vn - fn;
    } else {
      float vim1 = v[i-1];
      float vi   = v[i];
      float vip1 = v[i+1];
      float fi   = f[i];
      r[i]  = (vim1 - 2 * vi + vip1)*n2 - fi;
    }
    i += blockDim.x * gridDim.x;

  }

}

// Coarsens the grid.
__global__
void restrictCudaKernel(float *v2h, float *vh, const int nh) {

  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  while (i < nh/2) {
    if (i == 0) {
      v2h[0] = vh[0];
    } else {
      float vh_2im1 = vh[2*i-1];
      float vh_2i   = vh[2*i];
      float vh_2ip1 = vh[2*i+1];
      v2h[i] = (vh_2im1 + 2 * vh_2i + vh_2ip1) / 4;
    }
    i += blockDim.x * gridDim.x;
  }

}

// Refines the grid.
__global__
void interpolateCudaKernel(float *vh, float *v2h, const int n2h) {

  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  while (i < n2h) {
    float v2h_i   = v2h[i];
    float v2h_ip1 = v2h[i+1];
    vh[2*i] = v2h_i;
    vh[2*i+1] = (v2h_i + v2h_ip1) / 2;
    i += blockDim.x * gridDim.x;
  }

}

// Corrects numerical solution with computed errors.
__global__
void errorCorrectionCudaKernel(float *v, const float *e, const int n) {

  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  // I could have unrolled this loop (and many others), but I doubt
  // they were enough of a bottleneck in the code to matter
  // (especially this really simple one!)
  while (i < n+1) {
    v[i] -= e[i];
    i += blockDim.x * gridDim.x;
  }

}

// Iterates over the Jacobi kernel.
void relaxCuda(float *v, float *v0, const float *f, const int nu, const int n,
	       const unsigned int threadsPerBlock) {

  // shared memory parameter
  unsigned int size_sm = (threadsPerBlock + 2) * sizeof(float);
  unsigned int blocks = (int) ceil((float) n / threadsPerBlock);
  
  int it = 0;
  while (it < nu) {

    jacobiCudaKernel<<<blocks, threadsPerBlock, size_sm>>>( v, v0, f, n );

    float *tmp1 = v;
    v  = v0;
    v0 = tmp1;

    it++;
  }

  float *tmp2 = v;
  v  = v0;
  v0 = tmp2;
  
}


/* CYCLES:
 *
 * These are the cyclic solvers that manage changing grid sizes,
 * computing residuals and errors, and using these errors to correct
 * the initial guess (the input solution).
 */
void vCycleCuda(float *v, const float *f, const int L, int l,
		const unsigned int maxThreadsPerBlock) {

  const int n = pow(2, L-l);
  const int nu1 = 2;
  const int nu2 = 1;

  // Compute in this function
  unsigned int threadsPerBlock = min(maxThreadsPerBlock, n);
  unsigned int blocks = (int) ceil((float) n / threadsPerBlock);

  float *vh;
  hipMalloc( (void **) &vh, (n+1) * sizeof(float) );
  hipMemset( &vh[0], 0, sizeof(float) );
  hipMemset( &vh[n], 0, sizeof(float) );

  // Jacobi iteration filtering
  relaxCuda(vh, v, f, nu1, n, threadsPerBlock);
  hipMemset( &vh[0], 0, sizeof(float) );
  hipMemset( &vh[n], 0, sizeof(float) );

  if (l < L-1) {

    // Compute residual
    float *rh;
    hipMalloc( (void **) &rh, (n+1) * sizeof(float) );
    residualCudaKernel<<<blocks, threadsPerBlock>>>(rh, vh, f, n);

    // h -> 2h
    threadsPerBlock = min(n/2, maxThreadsPerBlock);
    blocks = (int) ceil((float) n/2 / threadsPerBlock);
    
    // Restrict residual
    float *r2h;
    hipMalloc( (void **) &r2h, (n/2+1) * sizeof(float) );
    restrictCudaKernel<<<blocks, threadsPerBlock>>>(r2h, rh, n);
    hipFree(rh);

    // Next subset of v-cycle: compute error
    float *e2h;
    hipMalloc( (void **) &e2h, (n/2+1) * sizeof(float) );
    hipMemset( e2h, 0, (n/2+1) * sizeof(float) );
    vCycleCuda(e2h, r2h, L, l+1, maxThreadsPerBlock);
    hipFree(r2h);
    
    // Interpolate error
    float *eh;
    hipMalloc( (void **) &eh, (n+1) * sizeof(float) );
    interpolateCudaKernel<<<blocks, threadsPerBlock>>>(eh, e2h, n/2);
    hipFree(e2h);

    // 2h -> h
    threadsPerBlock = min(n, maxThreadsPerBlock);
    blocks = (int) ceil((float) n / threadsPerBlock);
    
    // Error correction
    errorCorrectionCudaKernel<<<blocks, threadsPerBlock>>>(vh, eh, n);
    hipMemset(&vh[0], 0, sizeof(float));
    hipMemset(&vh[n], 0, sizeof(float));

  }

  // Jacobi iteration filtering
  relaxCuda(v, vh, f, nu2, n, threadsPerBlock);

  hipFree(vh);
}

void fCycleCuda(float *vh, float *fh, const int L, int l,
		const unsigned int maxThreadsPerBlock) {

  const int n = pow(2, L-l);
  const int nu0 = 1;

  // Compute in this function
  unsigned int blocks;
  unsigned int threadsPerBlock;
  
  if (l < L-1) {

    // h -> 2h
    threadsPerBlock = min(n/2, maxThreadsPerBlock);
    blocks = (int) ceil((float) n/2 / threadsPerBlock);

    // Coarsen input vh
    float *v2h;
    hipMalloc( (void **) &v2h, (n/2+1) * sizeof(float) );
    restrictCudaKernel<<<blocks, threadsPerBlock>>>(v2h, vh, n);
    hipMemset(&v2h[0], 0, sizeof(float));
    hipMemset(&v2h[n/2], 0, sizeof(float));

    // Coarsen forcing function
    float *f2h;
    hipMalloc( (void **) &f2h, (n/2+1) * sizeof(float) );
    restrictCudaKernel<<<blocks, threadsPerBlock>>>(f2h, fh, n);

    // Coarsen them again, recursively
    fCycleCuda(v2h, f2h, L, l+1, maxThreadsPerBlock);
    hipFree(f2h);

    // Refine grid for the NEXT V-cycle increment
    interpolateCudaKernel<<<blocks, threadsPerBlock>>>(vh, v2h, n/2);
    hipFree(v2h);
  }

  // V-cycle!
  int iter = 0;
  while (iter < nu0) {
    vCycleCuda(vh, fh, L, l, maxThreadsPerBlock);
    iter++;
  }
  
}


/* SOLVERS: Called from the host
 *
 * These set up the problem for cuda solvers (e.g. allocate memory),
 * and manage iterations, computing errors, measuring time, and
 * printing results.
 */
void jacobiSolverCuda(const float *f_host, const float *u_host, const int L,
		      const unsigned int threadsPerBlock) {

  const int n = pow(2, L);

  // Numerical solution
  float **v = new float*[2];
  hipMalloc( (void **) &v[0], (n+1) * sizeof(float) );
  hipMalloc( (void **) &v[1], (n+1) * sizeof(float) );
  hipMemset( v[0], 0, (n+1) * sizeof(float) );
  hipMemset( v[1], 0, (n+1) * sizeof(float) );

  // Forcing function
  float *f;
  hipMalloc( (void **) &f, (n+1) * sizeof(float) );
  hipMemcpy( f, f_host, (n+1) * sizeof(float), hipMemcpyHostToDevice );

  // Exact solution
  float *u;
  hipMalloc( (void **) &u, (n+1) * sizeof(float) );
  hipMemcpy( u, u_host, (n+1) * sizeof(float), hipMemcpyHostToDevice );

  // Error
  float *maxError;
  hipMalloc( (void **) &maxError, sizeof(float) );
  hipMemset( maxError, 0, sizeof(float) );

  // Other
  int iter = 0;
  float e_prev = 1;
  float *error = new float[1];
  error[0] = 0;
  unsigned int size_sm = (threadsPerBlock + 2) * sizeof(float);
  unsigned int blocks = (int) ceil( (float) n / threadsPerBlock );

  // Solver
  float tic = clock();
  while (abs(error[0]-e_prev) > 1e-4) {

    float *vprev = v[iter % 2];
    float *vcurr = v[(iter+1) % 2];

    jacobiCudaKernel<<<blocks, threadsPerBlock, size_sm>>>( vcurr, vprev, f, n );

    // Ensure BCs are 0
    hipMemset( &vcurr[0], 0, sizeof(float) );
    hipMemset( &vcurr[n], 0, sizeof(float) );

    if (iter % 1000 == 0) {

      // Store previous error
      e_prev = error[0];

      // Compute error
      size_sm = threadsPerBlock * sizeof(float);
      unsigned int blocks = (int) ceil((float) n / threadsPerBlock);
      computeErrorKernel<<<blocks, threadsPerBlock, size_sm>>>( u, vcurr, n, maxError);
      hipMemcpy( error, maxError, sizeof(float), hipMemcpyDeviceToHost );
    }

    iter++;
  }
  float toc = clock();
  float time = (toc - tic) / CLOCKS_PER_SEC;
  printf("%d\t%d\t%d\t%6.2e\t%f\t%6.2e\n", L, n, iter, error[0], time, time / iter);

  hipFree(v[0]);
  hipFree(v[1]);
  hipFree(f);
  hipFree(u);

}

void vCycleSolverCuda(const float *f_host, const float *u_host, const int L,
		      const unsigned int threadsPerBlock) {

  const int n = pow(2, L);

  // Numerical solution
  float *v;
  hipMalloc( (void **) &v, (n+1) * sizeof(float) );
  hipMemset( v, 0, (n+1) * sizeof(float) );

  // Forcint function
  float *f;
  hipMalloc( (void **) &f, (n+1) * sizeof(float) );
  hipMemcpy( f, f_host, (n+1) * sizeof(float), hipMemcpyHostToDevice );

  // Exact solution
  float *u;
  hipMalloc( (void **) &u, (n+1) * sizeof(float) );
  hipMemcpy( u, u_host, (n+1) * sizeof(float), hipMemcpyHostToDevice );

  // Error
  float *maxError;
  hipMalloc( (void **) &maxError, sizeof(float) );
  hipMemset( maxError, 0, sizeof(float) );

  // Other
  int iter = 0;
  float e_prev = 1;
  float *error = new float[1];
  error[0] = 0;
  unsigned int size_sm = threadsPerBlock * sizeof(float);
  unsigned int blocks = (int) ceil((float) n / threadsPerBlock);

  float tic = clock();
  while (abs(error[0]-e_prev) > 1e-15) {

    vCycleCuda(v, f, L, 0, threadsPerBlock);

    // Ensure BCs are 0
    hipMemset( &v[0], 0, sizeof(float) );
    hipMemset( &v[n], 0, sizeof(float) );

    // Store previous error
    e_prev = error[0];

    // Compute error
    computeErrorKernel<<<blocks, threadsPerBlock, size_sm>>>( u, v, n, maxError);
    hipMemcpy( error, maxError, sizeof(float), hipMemcpyDeviceToHost );

    iter++;
  }
  float toc = clock();
  float time = (toc - tic) / CLOCKS_PER_SEC;
  printf("%d\t%d\t%d\t%6.2e\t%f\t%6.2e\n", L, n, iter, error[0], time, time / iter);

  hipFree(v);
  hipFree(f);
  hipFree(u);

}


void fCycleSolverCuda(const float *f_host, const float *u_host, const int L,
		      const unsigned int threadsPerBlock) {

  const int n = pow(2, L);

  // Numerical solution
  float *v;
  hipMalloc( (void **) &v, (n+1) * sizeof(float) );
  hipMemset( v, 0, (n+1) * sizeof(float) );

  // Forcing function
  float *f;
  hipMalloc( (void **) &f, (n+1) * sizeof(float) );
  hipMemcpy( f, f_host, (n+1) * sizeof(float), hipMemcpyHostToDevice );

  // Exact solution
  float *u;
  hipMalloc( (void **) &u, (n+1) * sizeof(float) );
  hipMemcpy( u, u_host, (n+1) * sizeof(float), hipMemcpyHostToDevice );

  // Error
  float *maxError;
  hipMalloc( (void **) &maxError, sizeof(float) );
  hipMemset( maxError, 0, sizeof(float) );

  // Other
  unsigned int size_sm = threadsPerBlock * sizeof(float);
  unsigned int blocks = (int) ceil((float) n / threadsPerBlock);
  
  // Solver
  float tic = clock();
  fCycleCuda(v, f, L, 0, threadsPerBlock);

  // Error
  computeErrorKernel<<<blocks, threadsPerBlock, size_sm>>>( u, v, n, maxError);
  float *error = new float[1];
  hipMemcpy( error, maxError, sizeof(float), hipMemcpyDeviceToHost );

  float toc = clock();
  float time = (toc - tic) / CLOCKS_PER_SEC;
  printf("%d\t%d\t%d\t%6.2e\t%f\t%6.2e\n", L, n, 1, error[0], time, time);

  hipFree(v);
  hipFree(f);
  hipFree(u);
  
  
}
